#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

#include <lr.hh>
#include <kernel.cu>

using namespace std;
//lrGPU
lrGPU::lrGPU(float* data_in, float* labels_in, int points, int features, int iter, float a){

    npoints = points;
    nfeatures = features;
    maxIter = iter;
    alpha = a;

    data_bytes = npoints*(nfeatures+1)*sizeof(float);
    weight_bytes = (nfeatures+1)*sizeof(float);
    label_bytes = npoints*sizeof(float);

    if(data != NULL) hipFree(data);
    hipMallocManaged((void **)&data, data_bytes);
    hipMemcpy(data, data_in, data_bytes, hipMemcpyHostToDevice);

    float *dT = new float[npoints*(nfeatures+1)];
    for (int i = 0; i < npoints; i++){
        for (int j = 0; j < nfeatures+1; j++){
            dT[j*npoints + i] = data_in[j+i*(nfeatures+1)];
        }
    }

    if(dataT != NULL) hipFree(dataT);
    hipMallocManaged((void **)&dataT, data_bytes);
    hipMemcpy(dataT, dT, data_bytes, hipMemcpyHostToDevice);

    if(labels != NULL) hipFree(labels);
    hipMallocManaged((void **)&labels, label_bytes);
    hipMemcpy(labels, labels_in, label_bytes, hipMemcpyHostToDevice);

    float* randWeights = new float[nfeatures+1];

    for (int i = 0; i < nfeatures+1; i++){
        randWeights[i] = ((double)rand() / (RAND_MAX));
    }

    if(weights != NULL) hipFree(weights);
    hipMallocManaged((void **)&weights, weight_bytes);
    hipMemcpy(weights, randWeights, weight_bytes, hipMemcpyHostToDevice);

    blocksize = 32;
    blkDim = dim3(1, blocksize*blocksize, 1);
    grdDim = dim3(1, BLK(npoints, blocksize*blocksize), 1);
    MMBlkDim = dim3(blocksize, blocksize, 1);
    MMGrdDim = dim3(1, BLK(npoints, blocksize), 1);
    MMTGrdDim = dim3(1, BLK(nfeatures, blocksize), 1);

}

lrGPU::~lrGPU()
{
    Cleanup();
}

void lrGPU::Cleanup()
{
    if(data != NULL) hipFree(data);
    if(labels != NULL) hipFree(labels);
    if(weights != NULL) hipFree(weights);
}

void lrGPU::run()
{
    float *error, *prob;
    hipMallocManaged((void **)&error, label_bytes);
    hipMallocManaged((void **)&prob, label_bytes);
    float *oldWeights; hipMallocManaged((void **)&oldWeights, weight_bytes);
    float *change; hipMallocManaged((void **)&change, weight_bytes);
    float tol = 0.0001;
    float *check = new float[nfeatures+1];
    float sum = 0.0;
    int grid = BLK(nfeatures, blocksize);

    for (int i = 0; i < maxIter; i++){
		
		hipMemcpy(oldWeights, weights, weight_bytes, hipMemcpyDeviceToDevice);
		sum = 0.0;
	
		MatrixMultiplyKernel<<<MMGrdDim, MMBlkDim>>>(data, weights, prob, npoints, 1, nfeatures + 1, 1.0, 0.0);
		sub_sigKernel<<<grdDim, blkDim>>>(labels, prob, error, npoints);
		MatrixMultiplyKernel <<<MMTGrdDim, MMBlkDim>>>(dataT, error, weights, nfeatures + 1, 1, npoints, alpha, 1.0);
		
		distKernel<<<dim3(1,grid,1), dim3(1, blocksize, 1)>>>(weights, oldWeights, change, nfeatures+1);
		hipMemcpy(check, change, weight_bytes, hipMemcpyDeviceToHost);
		
		for(int j = 0; j < nfeatures + 1; j++){
			sum += check[j];
		}	
		sum = sqrt(sum);
		alpha = alpha - alpha/(maxIter - i);
		
		if(sum < (tol*(nfeatures+1))){
			cout << i+1 << " iterations" << endl;
			break;
		}
        }
    cout << sum << " final dist" << endl;

    hipFree(error); hipFree(prob); hipFree(oldWeights); hipFree(change);
}

int lrGPU::classify(float* point_in)
{
    //Use classify to classify the point
    float classify = 0.0;
    float *updatedWeights;

    //Copy the weights back to the host
    hipMallocManaged((void **)&updatedWeights, (nfeatures+1)*sizeof(float));
    hipMemcpy(updatedWeights, weights, (nfeatures + 1)*sizeof(float), hipMemcpyDeviceToHost);

    //Classify the point by summing the products of corresponding weights and data dimensions
    for (int k = 0; k < nfeatures + 1; k++){
        classify += updatedWeights[k] * point_in[k];
    }

    //Classify the point
    classify = sigmoid(classify);
    if (classify > 0.5){
        return 1;
    }else{
        return 0;
    }
}
